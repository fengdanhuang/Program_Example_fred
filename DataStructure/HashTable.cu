#include "hip/hip_runtime.h"


#define SIZE    (100*1024*1024) 
#define ELEMENTS    (SIZE / sizeof(unsigned int)) 
#define HASH_ENTRIES     1024

struct Entry {
	unsigned int    key;
	void*           value;
    Entry           *next; 
};


struct Table {
	size_t  count;
    Entry   **entries;
    Entry   *pool; 
}; 


__device__ __host__ size_t hash( unsigned int value, size_t count ) {
	return value % count; 
}


void initialize_table( Table &table, int entries, int elements ) {
    table.count = entries;
    hipMalloc( (void**)&table.entries, entries * sizeof(Entry*));
    hipMemset( table.entries, 0, entries * sizeof(Entry*) );
    hipMalloc( (void**)&table.pool, elements * sizeof(Entry)); 
}


void free_table( Table &table ) {
    hipFree( table.pool );
    hipFree( table.entries ); 
}


void verify_table( const Table &dev_table ) {
    Table   table;
    copy_table_to_host( dev_table, table );
	int count = 0;
	for (size_t i=0; i<table.count; i++) {
        Entry   *current = table.entries[i];
		while (current != NULL) {
            ++count;
			if (hash( current->value, table.count ) != i){
				printf( "%d hashed to %ld, but was located at %ld\n", current->value, hash(current->value, table.count), i );
			}	
			current = current->next;
        }
    }
	if (count != ELEMENTS) printf( ��%d elements found in hash table.  Should be %ld\n��, count, ELEMENTS );
	else printf( ��All %d elements found in hash table.\n��, count );
    free( table.pool );
    free( table.entries ); 
}


void copy_table_to_host( const Table &table, Table &hostTable) {
    hostTable.count = table.count;
    hostTable.entries = (Entry**)calloc( table.count, sizeof(Entry*) );
    hostTable.pool = (Entry*)malloc( ELEMENTS * sizeof( Entry ) );
    hipMemcpy( hostTable.entries, table.entries, table.count * sizeof(Entry*), hipMemcpyDeviceToHost );
    hipMemcpy( hostTable.pool, table.pool, ELEMENTS * sizeof( Entry ), hipMemcpyDeviceToHost);
	for (int i=0; i<table.count; i++) {
		if (hostTable.entries[i] != NULL){
            hostTable.entries[i] = (Entry*)((size_t)hostTable.entries[i] - (size_t)table.pool + (size_t)hostTable.pool);
		}
    }
	for (int i=0; i<ELEMENTS; i++) {
		if (hostTable.pool[i].next != NULL){
            hostTable.pool[i].next = (Entry*)((size_t)hostTable.pool[i].next - (size_t)table.pool + (size_t)hostTable.pool);
		}	
    } 
}


__global__ void add_to_table( unsigned int *keys, void **values, Table table, Lock *lock ) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while (tid < ELEMENTS) {
		unsigned int key = keys[tid];
		size_t hashValue = hash( key, table.count );
		for (int i=0; i<32; i++) {
			if ((tid % 32) == i) {
                Entry *location = &(table.pool[tid]);
                location->key = key;
                location->value = values[tid];
                lock[hashValue].lock();
                location->next = table.entries[hashValue];
                table.entries[hashValue] = location;
                lock[hashValue].unlock();
            }
        }
        tid += stride;
    } 
}


int main( int argc, char** argv ) {
	unsigned int *buffer = (unsigned int*)big_random_block( SIZE );
    hipEvent_t     start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop );
    hipEventRecord( start, 0 );
	unsigned int* dev_keys;
	void** dev_values;
    hipMalloc( (void**)&dev_keys, SIZE );
    hipMalloc( (void**)&dev_values, SIZE );
    hipMemcpy( dev_keys, buffer, SIZE, hipMemcpyHostToDevice );
    // copy the values to dev_values here, filled in by user of this code example
    Table table;
    initialize_table( table, HASH_ENTRIES, ELEMENTS );
	Lock lock[HASH_ENTRIES];
    Lock *dev_lock;
    hipMalloc( (void**)&dev_lock, HASH_ENTRIES * sizeof( Lock ) );
    hipMemcpy( dev_lock, lock, HASH_ENTRIES * sizeof( Lock ), hipMemcpyHostToDevice );
	
	add_to_table<<<60,256>>>( dev_keys, dev_values, table, dev_lock );
	
    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
	float   elapsedTime;
    hipEventElapsedTime( &elapsedTime, start, stop );
    printf( "Time to hash:  %3.1f ms\n", elapsedTime );
    verify_table( table );
    hipEventDestroy( start );
    hipEventDestroy( stop );
    free_table( table );
    hipFree( dev_lock );
    hipFree( dev_keys );
    hipFree( dev_values );
    free( buffer );
	return 0; 
}