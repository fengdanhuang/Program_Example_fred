
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

__global__ void add(int a, int b, int *c) {
	*c = a + b;
	//must compiled under compiler:cuda4.0 or above, runned under Fermi architecture
	//eg /opt/cuda42/bin/nvcc -arch sm_20 page25_sum.cu
	printf("I am inside.\n");
}

int main (void){
	int c;
	int *dev_c;
	hipMalloc ((void**)&dev_c, sizeof(int));

	add<<<1,1>>>(2, 7, dev_c);
	
        hipMemcpy (&c,
			 dev_c,
			 sizeof(int),
			 hipMemcpyDeviceToHost );
	printf( "2 + 7 = %d\n", c );
	hipFree( dev_c );

	return 0;
}

