#include <stdio.h>
//#include <hip/hip_runtime.h>

int main(void) {
	hipDeviceProp_t prop;//cudaDevieProp is a structure, prop is a structure variable.
	
	int count;
	hipGetDeviceCount (&count);
	for (int i=0; i< count; i++){
		hipGetDeviceProperties (&prop, i);
		printf("\n****************************Device %d*************************************\n",i);
		printf("----------General Information----------\n");
		printf("Name: %s\n", prop.name);
		printf("Compute capability: %d.%d\n", prop.major, prop.minor);
	//	printf("Clock rate: %d\n", prop.clockRate);
		printf("Clock rate: %d MHz\n", (prop.clockRate/1000));

		printf("GPU type: ");
		if (prop.integrated)
			printf ("Integrated GPU\n");
		else
			printf ("Discreted GPU\n");

		printf("Device copy overlap: ");		
		if (prop.deviceOverlap)	
			printf ("Enabled\n");
		else
			printf ("Disabled\n");

		printf( "Kernel execution timeout: ");		
		if (prop.kernelExecTimeoutEnabled)
			printf ("Enabled\n");
		else
			printf ("Disabled\n");

		printf( "Mapping Host Memory into CUDA Device Address Space: ");
		if (prop.canMapHostMemory)
			printf ("Enabled\n");
		else
			printf ("Disabled\n");

		printf( "Executing Multiple Kernels Support: ");
		if (prop.concurrentKernels)
			printf ("Enabled\n");
		else 
			printf ("Disabled\n");

		printf( "Device Computing Mode: ");
		if (prop.computeMode == 0) printf("Default\n");
		if (prop.computeMode == 1) printf("Exclusive\n");
		if (prop.computeMode == 2) printf("Prohibited\n");
			
		printf( "\n----------Memory Information for device----------\n");
	//	printf( "Total Global Memory: %ld\n", prop.totalGlobalMem);
		printf( "Total Global Memory: %ld MB\n", (prop.totalGlobalMem/1024/1024));
	//	printf(	"Total Constant Memory: %ld\n", prop.totalConstMem );
		printf( "Total Constant Memory: %ld kB\n", (prop.totalConstMem/1024));	
	//	printf( "Total Constant Memory: %ld MB\n", (prop.totalConstMem/1024/1024));
	//	printf( "Max Memory Pitch Allowed for Memory Copies: %ld\n", prop.memPitch );
		printf( "Max Memory Pitch Allowed for Memory Copies: %ld MB\n", (prop.memPitch/1024/1024));
		printf( "Texture Alignment: %ld B\n", prop.textureAlignment );
		printf( "Max Size Supported for 1D Textures: %ld\n", prop.maxTexture1D);
		printf( "Max Dimensions Supported for 2D Texture: (%d, %d)\n", prop.maxTexture2D[0], prop.maxTexture2D[1]);
		printf( "Max Dimensions Supported for 3D Texture: (%d, %d, %d)\n", 
										prop.maxTexture3D[0],
										prop.maxTexture3D[1],
										prop.maxTexture3D[2]);
/*		printf( "Max Dimensions Supported for 2D Texture Array: (%d, %d, %d)\n", 
										prop.maxTexture2DArray[0],
										prop.maxTexture2DArray[1],
										prop.maxTexture2DArray[2]);
*/		
		printf( "\n----------Thread Information for device----------\n");
		printf( "Streaming Multiprocessor count: %d\n", prop.multiProcessorCount);
	//	printf( "Shared Memory per Block: %ld\n", prop.sharedMemPerBlock);
		printf( "Shared Memory per Block: %ld KB\n", (prop.sharedMemPerBlock/1024));
		printf( "Registers per Block: %ld\n", prop.regsPerBlock);
		printf( "Threads in warp: %ld\n", prop.warpSize);
		printf( "Max threads per block: %d\n",
					prop.maxThreadsPerBlock);			
		printf( "Max thread dimensions: (%d, %d, %d)\n",
					prop.maxThreadsDim[0], prop.maxThreadsDim[1],
					prop.maxThreadsDim[2]);
		printf( "Max grid dimensions: (%d, %d, %d)\n",
					prop.maxGridSize[0], prop.maxGridSize[1],
					prop.maxGridSize[2]);
		printf("*****************************************************************\n");
		}
	
	return 0;
}

