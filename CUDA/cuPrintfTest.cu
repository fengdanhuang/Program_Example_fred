#include "hip/hip_runtime.h"
#include <stdio.h>  
#include <stdlib.h>  
#include <time.h>  
#include <hip/hip_runtime.h>  
#include "cuPrintf.cu"  
bool InitGPUSet()  
{  
    char GPU[100] = "GPU: ";  
    hipDeviceProp_t tCard;  
    int num = 0;  
    if(hipSuccess == hipGetDeviceCount(&num))  
    {  
        for(int i = 0; i < num; ++ i)  
        {  
            hipSetDevice(i);  
            hipGetDeviceProperties(&tCard, i);  
            puts(strcat(GPU , tCard.name));//返回的就是链接后的结果,也为其的嵌套使用提供了条件  
        }  
    }  
    else  return false;  
    return true;  
}  
bool cuPrintInit()  
{  
    hipError_t err = cudaPrintfInit();  
    if(0 != strcmp("no error", hipGetErrorString(err)))  return false;  
    return true;  
}  
__global__ void displayGPU_demo()  
{  
    int bsize = blockDim.x;  
    int bid = blockIdx.x;  
    int tid = bid * bsize + threadIdx.x;  
    cuPrintf("当前执行kernel的 block 编号:/t%d\n", bid);  
    cuPrintf("当前执行kernel的 thread 在当前块中编号:/t%d\n", threadIdx.x);  
    cuPrintf("当前执行kernel的 thread 全局编号:/t%d\n", tid);  
    cuPrintf("thread over\n\n");  
}  

int main(void)  
{  
    if(!InitGPUSet())  puts("device is not ready!");  
    else if(!cuPrintInit())  puts("device is not ready!");  
    else  
    {  
        displayGPU_demo<<<2, 3>>>();  
        cudaPrintfDisplay(stdout, true);//true输出是哪一个block的第几个thread在执行本条输出语句，形如：[blockID, threadID]；false不输出  
        cudaPrintfEnd();  
    }  
    return 0;  
}  
