#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include "cuPrintf.cu"

#define N 2

__global__ void add2D(int **d_x, int **d_y, int **d_sum){

	//int tid = blockIdx.y*N+blockIdx.x;
	
	d_sum[threadIdx.y][threadIdx.x]=d_x[threadIdx.y][threadIdx.x]+d_y[threadIdx.y][threadIdx.x];
	cuPrintf("d_sum[%d][%d]=%d\n",threadIdx.y, threadIdx.x, d_sum[threadIdx.y][threadIdx.x]);	
}

int main(){
	int x[N][N], y[N][N], sum[N][N];
	int **d_x, **d_y, **d_sum;
	size_t dpitch_x=N, dpitch_y=N, dpitch_sum=N;
//	size_t dpitch;
	dim3 blockDim(N,N,1);
	for(int i=0;i<N;i++){
		for(int j=0;j<N;j++){
			x[i][j]=3;y[i][j]=6;
		}
	}
	
	cudaPrintfInit();

	hipMallocPitch((void**)&d_x, &dpitch_x, N*sizeof(int), N);
	hipMallocPitch((void**)&d_y, &dpitch_y, N*sizeof(int), N);
	hipMemcpy2D(d_x,dpitch_x, x, N*sizeof(int), N*sizeof(int),N,hipMemcpyHostToDevice);
	hipMemcpy2D(d_y,dpitch_y, y, N*sizeof(int), N*sizeof(int),N,hipMemcpyHostToDevice);
	
	add2D<<<1,blockDim>>>(d_x,d_y,d_sum);
	cudaPrintfDisplay(stdout,true);
	cudaPrintfEnd();
	hipMemcpy2D(sum,N*sizeof(int), d_sum, dpitch_sum, N*sizeof(int),N,hipMemcpyDeviceToHost);
/*	for(int i=0;i<N;i++){
		for(int j=0;j<N;j++){
			printf("sum[%d][%d]=%d ",i,j,sum[i][j]);
		}
		printf("\n");
	}*/
	return 0;
}






	
