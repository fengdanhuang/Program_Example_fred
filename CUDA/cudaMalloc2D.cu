#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>  
#include <time.h>  
#include <hip/hip_runtime.h> 
#include "cuPrintf.cu"


bool InitGPUSet();
bool cuPrintInit();
__global__ void MyKernel(float* devPtr, size_t pitch, int width, int height);

int main(){
	int width = 5, height = 5;
	float *devPtr;
	size_t pitch = 10;
	
//	gridDim.x=2;gridDim.y=2;gridDim.z=2;
//	blockDim.x=2;blockDim.y=2;blockDim.z=2;

	dim3 gridDim(2,2,3);
	dim3 blockDim(2,2,2);

	if(!cuPrintInit())  puts("device is not ready!");
	printf("pitch before:%d\n",pitch);
	hipMallocPitch(&devPtr, &pitch, width*sizeof(float), height);
	MyKernel<<<gridDim,blockDim>>>(devPtr,pitch,width,height);
	cudaPrintfDisplay(stdout, true);//true输出是哪一个block的第几个thread在执行本条输出语句，形如：[blockID, threadID]；false不输出  
        cudaPrintfEnd(); 
	hipFree(devPtr);
	printf("pitch after:%d\n", pitch);

	return 0;
}


__global__ void MyKernel(float* devPtr, size_t pitch, int width, int height){
	cuPrintf("当前执行kernel的 block 编号:/t%d\n");
	cuPrintf("width=%d,height=%d,pitch=%d\n", width, height,pitch);
	cuPrintf("gridDim.x=%d,gridDim.y=%d,gridDim.z=%d\n", gridDim.x, gridDim.y,gridDim.z);
	cuPrintf("blockDim.x=%d,blockDim.y=%d,blockDim.z=%d\n", blockDim.x, blockDim.y, blockDim.z);
	//cuPrintf("blockIdx.x=%d, blockIdx.y=%d\n", blockIdx.x, blockIdx.y,blockIdx.z);
	//cuPrintf("threadIdx.x=%d, threadIdx.y=%d, threadIdx.z=%d\n",threadIdx.x,threadIdx.y,threadIdx.z);
	cuPrintf("blockIdx.z=%d, blockIdx.y=%d,blockIdx.x=%d\n", blockIdx.z, blockIdx.y,blockIdx.x);
	cuPrintf("threadIdx.z=%d, threadIdx.y=%d, threadIdx.x=%d\n",threadIdx.z,threadIdx.y,threadIdx.x);
	for(int r=0;r<height;++r){
		float *row = (float*)((char*)devPtr+r*pitch);
		for(int c=0; c<width; ++c){
			float element = row[c];
			cuPrintf("%f ",element);
		}
		cuPrintf("\n");
	}
	cuPrintf("\n");
}

bool InitGPUSet()  
{  
    char GPU[100] = "GPU: ";  
    hipDeviceProp_t tCard;  
    int num = 0;  
    if(hipSuccess == hipGetDeviceCount(&num))  
    {  
        for(int i = 0; i < num; ++ i)  
        {  
            hipSetDevice(i);  
            hipGetDeviceProperties(&tCard, i);  
            puts(strcat(GPU , tCard.name));//返回的就是链接后的结果,也为其的嵌套使用提供了条件  
        }  
    }  
    else  return false;  
    return true;  
}  

bool cuPrintInit()  {  
    hipError_t err = cudaPrintfInit();  
    if(0 != strcmp("no error", hipGetErrorString(err)))  return false;  
    return true;  
}  
