#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__global__ void MyKernel(float** dev_matrix, size_t pitch, int width, int height)
{   
        int number = 0;
        for (int i = 0; i < height; ++i) 
        {
                float* row = (float*)((char*)dev_matrix + i*pitch);

                for (int j = 0; j < width; ++j) 
                {       
                        row[j] = number; 
                        number++;
                }
        }
}


int main (int argc , char * argv [])
{
        int width = 4, height = 2, i, j;
//      float matrix[width][height];
	float matrix[height][width];
        float **dev_matrix;
        size_t pitch;

    	printf("\nMATRIX MANIPULATION\n");

        for (i = 0; i < height; i++) 
                for (j = 0; j < width; j++)
                        matrix[i][j] = 0.0;
 
        printf("Matrix in host memory\n");
        for (i = 0; i < height; i++) 
        {       
                for (j = 0; j < width; j++)
                        printf("%f   ", matrix[i][j]);
                printf("\n");
        }

        hipMallocPitch(&dev_matrix, &pitch, width * sizeof(float), height);

        hipMemcpy2D(dev_matrix, pitch, matrix, width * sizeof(float), width * sizeof(float), height, hipMemcpyHostToDevice);  

        MyKernel<<<2, 2>>>(dev_matrix, pitch, width, height);

        hipMemcpy2D(matrix, width * sizeof(float), dev_matrix, pitch, width * sizeof(float), height, hipMemcpyDeviceToHost);

        printf("Matrix after calculate elements in the gpu\n");
        for (i = 0; i < height; i++) 
        {       
                for (j = 0; j < width; j++)
                        printf("%f   ", matrix[i][j]);
                printf("\n");
        }

    hipFree(dev_matrix);

        return 0;
}
